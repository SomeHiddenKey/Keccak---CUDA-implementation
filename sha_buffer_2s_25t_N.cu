
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <iostream>
#include <fstream>
#include <sstream> 
#include <chrono>
#include <thread> 

#define LOGG true
#define BUFFER_SIZE 400

typedef unsigned char uint8_t;

typedef char* string_t;

#define total 1600
#define rate 1088

#define rounds 24

#define totalBytes total/8
#define rateBytes rate/8
#define rateBytesBuffer rateBytes*BUFFER_SIZE
#define rateLanes rate/64

#define delim_begin 0x06
#define delim_end 0x80

#define ROT(a, offset) ((((uint64_t)a) << offset) ^ (((uint64_t)a) >> (64-offset))) //credit
#define state (*State)
#define lane ((uint64_t*)state._8)

#define CROSSPLANE(result,x, array) result[x] = (array[x] ^ array[x + 5] ^ array[x + 10] ^ array[x + 15] ^ array[x + 20])
#define OMEGA(cross, result, m, i) result[i] ^= (cross[m==0?4:(m-1)] ^ ROT(cross[m==4?0:(m+1)], 1))
#define RHOPI(result, array, index, i,rho) \
{ index = pi[i];  \
result[i] = ROT(array[indx], rho[indx]); \
}
#define CHI(result, array, i, z) result[i] = array[i] ^ (~array[(i+1)%5 +z*5] & array[(i+2)%5 +z*5]);
#define IOTA(result, iota, round) result[0] ^= iota[round];

__device__ __constant__ uint8_t rho[25] =
    {0, 1, 62, 28, 27,
     36, 44, 6, 55, 20,
     3, 10, 43, 25, 39,
     41, 45, 15, 21, 8,
     18, 2, 61, 56, 14};

__device__ __constant__ uint8_t pi[25] =
    {0, 6, 12, 18, 24,
     3, 9, 10, 16, 22,
     1, 7, 13, 19, 20,
     4, 5, 11, 17, 23,
     2, 8, 14, 15, 21};

__device__ __constant__ uint64_t iota[24] =
  {
    0x0000000000000001UL, 0x0000000000008082UL,0x800000000000808aUL, 0x8000000080008000UL,
    0x000000000000808bUL, 0x0000000080000001UL,0x8000000080008081UL, 0x8000000000008009UL,
    0x000000000000008aUL, 0x0000000000000088UL,0x0000000080008009UL, 0x000000008000000aUL,
    0x000000008000808bUL, 0x800000000000008bUL,0x8000000000008089UL, 0x8000000000008003UL,
    0x8000000000008002UL, 0x8000000000000080UL,0x000000000000800aUL, 0x800000008000000aUL,
    0x8000000080008081UL, 0x8000000000008080UL,0x0000000080000001UL, 0x8000000080008008UL
};

union INTER{
    uint64_t _64[25];
    uint8_t _8[200];
};

__global__ void Keccak(string_t input, int size, union INTER* State){
    int buffer_i = 0;
    while(size>0 && (buffer_i++) < BUFFER_SIZE){
        if(size<rateBytes){    
            for(int i=0; i<size; i++)
                state._8[i] ^= input[i];
            state._8[size] ^= delim_begin;//padding
            state._8[rateBytes - 1] ^= delim_end;
        }
        else{
            for(int i=0; i<rateLanes; i++){
                state._64[i] ^= ((uint64_t*)input)[i];
            }   
            input += rateBytes;
        }
        
        __shared__ uint64_t CrossPlane[5];
        __shared__ union INTER iState;

        uint8_t i = threadIdx.x;
        uint8_t m = i%5;
        uint8_t z = i/5;
        int indx = 0;

        for(int round = 0; round<rounds; round++){
            //Omega
            //CrossPlane[m] = state._64[m] ^ state._64[m + 5] ^ state._64[m + 10] ^ state._64[m + 15] ^ state._64[m + 20];
            CROSSPLANE(CrossPlane,m,state._64);

            //state._64[i] ^= CrossPlane[m==0?4:(m-1)] ^ ROT(CrossPlane[m==4?0:(m+1)], 1);
            OMEGA(CrossPlane, state._64, m, i);
            
            //Rho and Pi
            //indx = pi[i];
            //iState._64[i] = ROT(state._64[indx], rho[indx]);
            RHOPI(iState._64, state._64, indx, i, rho);
            
            //Chi
            //state._64[i] = iState._64[i] ^ (~iState._64[(i+1)%5 +z*5] & iState._64[(i+2)%5 +z*5]);
            CHI(state._64, iState._64, i, z);

            //state._64[0] ^= iota[round];
            IOTA(state._64, iota, round);
        }

        size -= rateBytes;
    }
}

std::ofstream myfile;

void hostr(const char* path, int times=1){
    for(int tm = 0; tm<times; tm++){
        std::streamsize size;// = message2.length();
        std::streamsize sizeInit;
        char* contents = new char[rateBytesBuffer];
        std::ifstream istr(path, std::ios::in | std::ios::binary);
        std::streambuf* pbuf = NULL;

        if (istr) 
        {
            pbuf = istr.rdbuf();
            size = pbuf->pubseekoff(0, istr.end);
            sizeInit = size;
            std::cout << " File size is: " << size << "\n";
            pbuf->pubseekoff(0, istr.beg);       // rewind 
        }
        unsigned char output[32] = { "" };

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        char* input_d1;
        hipMalloc((void**)&input_d1, rateBytesBuffer);

        char* input_d2;
        hipMalloc((void**)&input_d2, rateBytesBuffer);

        union INTER* State_d;
        hipMalloc((void**)&State_d, sizeof(union INTER));
        hipMemset(State_d, 0, 200);

        hipStream_t stream_c;
        hipStream_t stream_k;
        hipStreamCreate(&stream_c);
        hipStreamCreate(&stream_k);
        
        int itter = 0;
        double hashed = pbuf->sgetn(contents, rateBytesBuffer);
        hipMemcpyAsync((itter?input_d2:input_d1), contents, size>rateBytesBuffer?rateBytesBuffer:size, hipMemcpyHostToDevice, stream_c);
        size -= rateBytesBuffer;

        for(int offset = 0; size>0; itter = !itter)
        {
            //double percentage = hashed / (double)sizeInit * 100.0;
            //std::cout << " Percentage done: " << percentage << "\r";
            Keccak<<<1,25,0,stream_k>>>((itter?input_d2:input_d1), size+rateBytesBuffer, State_d);
            size -= rateBytesBuffer;
            hashed += pbuf->sgetn(contents, rateBytesBuffer);
            hipMemcpyAsync((itter?input_d1:input_d2), contents, rateBytesBuffer, hipMemcpyHostToDevice, stream_c);  
            hipStreamSynchronize(stream_k);
        }
        Keccak<<<1,25,0,stream_c>>>((itter?input_d2:input_d1), size+rateBytesBuffer, State_d); 
        hipMemcpyAsync(output, State_d, 32 * sizeof(char), hipMemcpyDeviceToHost, stream_c); 
        hipStreamSynchronize(stream_c);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        printf(" %s - ", path);
        std::cout << "elapsed time: " << milliseconds/1000 << "s\n";
        if(LOGG)myfile << path << "; " << milliseconds/1000 << "\n" ;

        for (int i = 0; i < 32; i++)
            printf("%02x", output[i]);

        hipFree(input_d1);
        hipFree(input_d2);
        hipFree(State_d);
        printf("\n\n");
        istr.close();
    }
}

int main( int argc, char *argv[] ){
    if(LOGG)myfile.open ("b_2s25t.csv");
    for(int pt = 1; pt<argc; pt++)
        hostr(argv[pt],1);
    if(LOGG)myfile << "\n" ;
    if(LOGG)myfile.close();
}